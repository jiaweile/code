#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define N 10240000
#define ThreadPerBlock 128
#define NSTREAM 4

__global__ void multiply(double * a, double *b , double * output, int length)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid < length)
		output[tid] = a[tid] + b[tid];
}

int main()
{	
	hipSetDevice(1);
	int nbytes = N * sizeof (double);
	double *a, *b, *c;
	double *dev_A, *dev_B, *dev_C;
	a = (double *) malloc( N*sizeof(double));
	b = (double *) malloc( N*sizeof(double));
	c = (double *) malloc( N*sizeof(double));
	if(a == NULL){
		printf("Error malloc \n");
		exit(0);
	}
	int i; 
	for (i = 0; i < N ; i++)
	{
		a[i] = i;
		b[i] = i;
	}
	
	assert(hipMalloc((void**) &dev_A, nbytes) == hipSuccess);
	assert(hipMalloc((void**) &dev_B, nbytes) == hipSuccess);
	assert(hipMalloc((void**) &dev_C, nbytes) == hipSuccess);
	
	assert(hipMemcpy(dev_A, a, nbytes, hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(dev_B, b, nbytes, hipMemcpyHostToDevice) == hipSuccess);
	
	struct timeval begin, end;
	gettimeofday(&begin, NULL);
	
	int nblock = N/ThreadPerBlock;
	if ( N % ThreadPerBlock) nblock ++;
	multiply<<<nblock, ThreadPerBlock>>>(dev_A, dev_B, dev_C,N);
	assert( hipDeviceSynchronize()  == hipSuccess ) ;
	
	gettimeofday(&end, NULL);
	double time = 1000000*(end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec);
	printf("One Stream time: %lf ms \n", time);
	
	hipStream_t stream[NSTREAM];
	
	int n = N/NSTREAM;
	printf("%d\n", n);
	assert( hipDeviceSynchronize()  == hipSuccess ) ;
	
	gettimeofday(&begin, NULL);
	for (i = 0; i < NSTREAM; i++)
	{
		nblock = n/ThreadPerBlock;
		if(n % ThreadPerBlock) nblock++;
		assert(hipStreamCreate(&stream[i])== hipSuccess);
		multiply<<<nblock, ThreadPerBlock, 0, stream[i]>>>(&dev_A[i*n], &dev_B[i*n], &dev_C[i*n], n);
		assert(hipStreamDestroy(stream[i])== hipSuccess);
	}
	assert( hipDeviceSynchronize()  == hipSuccess ) ;
	
	gettimeofday(&end, NULL);
	time = 1000000*(end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec);
	printf("%d Stream time: %lf ms \n",NSTREAM, time);

	assert(hipMemcpy(c, dev_C, nbytes, hipMemcpyDeviceToHost) == hipSuccess);

	for (i = 0; i < N; i++)
	{
		int d = (int) c[i];
		int e = 2*i;
		if( d != e)
		{
			printf("Error, %d, %lf\n", i, c[i]);
			exit(0);
		}
	}
	printf("Passed!!\n");
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	
	free(a);
	free(b);
	free(c);
}

	
