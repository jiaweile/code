#include <stdio.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"


/* Define complex multiply operation */
__device__ hipComplex ComplexMul(hipComplex a, hipComplex b)
{
    hipComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;

}

/* Define CUDA kernel that squares the input complex array */
__global__ void  square_complex(hipComplex *in, hipComplex *out, int N)
{
 unsigned int index   = blockIdx.x*blockDim.x+threadIdx.x;
 if( index<N ) 
  {
   out[index] = ComplexMul(in[index], in[index]);
  }

}


/* 
   Fortran subroutine arguments are passed by references.   
   call fun( array_a, array_b, N) will be mapped to
   function (*a, *b, *N);
*/
extern "C" void cudafunction_(hipComplex *a, hipComplex *b,  int *Np)
{
  int block_size=4;
  hipComplex *a_d;
  int N=*Np;
 
  /* Allocate complex array on device */
  hipMalloc ((void **) &a_d , sizeof(hipComplex)*N);
  
  /* Copy array from host memory to device memory */
  hipMemcpy( a_d, a,  sizeof(hipComplex)*N   ,hipMemcpyHostToDevice);

  /* Compute execution configuration */
   dim3 dimBlock(block_size);
   dim3 dimGrid (N/dimBlock.x);
   if( N % block_size != 0 ) dimGrid.x+=1;

  /* Execute the kernel */
  square_complex<<<dimGrid,dimBlock>>>(a_d,a_d,N);

  /* Copy the result back */
   hipMemcpy( b, a_d, sizeof(hipComplex)*N,hipMemcpyDeviceToHost);  

  /* Free memory on the device */
  hipFree(a_d);

  return;
}

